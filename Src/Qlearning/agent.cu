#include "hip/hip_runtime.h"
/*************************************************************************
/* ECE 285: GPU Programmming 2019 Winter quarter
/* Author and Instructer: Cheolhong An
/* Copyright 2019
/* University of California, San Diego
/*************************************************************************/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "agent.h"

__device__ float *d_qtable;
__device__ int   *d_action;

// epsilon
__device__ float epsilon;

// qLearning Paramters
__device__ float learningRate;		// discount factor
__device__ float gradientDec;		// learning Rate alpha

// TODO: implement the following

__global__ void agentsInit(int *d_agentsActions, int size) {
	for (int i = 0; i < size; ++i) {
		d_agentsActions[i] = 0;
	}
}

__global__ void qtableInit(float *d_qtable, int size) {
	for (int i = 0; i < size; ++i) {
		d_qtable[i] = 0;
	}
}

__global__ void agentsUpdate(int2* cstate, int2* nstate, float *rewards) {

}

__global__ void updateEpsilon() {
	epsilon -= 0.1f;
}

void initGlobalVariables() {
	float ep = 1.0;
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(epsilon), &ep, sizeof(float)));

	float lr = 0.1;
	float gd = 0.2;
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(learningRate), &lr, sizeof(float)));
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(gradientDec), &gd, sizeof(float)));
}

float decEpsilon() {
	updateEpsilon << <1, 1 >> >();
	float h_epsilon = 0.0f;
	CHECK(hipMemcpyFromSymbol(&h_epsilon, HIP_SYMBOL(epsilon), sizeof(float)));
	return h_epsilon;
}