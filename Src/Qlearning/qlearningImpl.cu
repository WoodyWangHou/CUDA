#include "hip/hip_runtime.h"
/*************************************************************************
/* ECE 285: GPU Programmming 2019 Winter quarter
/* Author and Instructer: Cheolhong An
/* Copyright 2019
/* University of California, San Diego
/*************************************************************************/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "agent.h"
#include "qlearningImpl.h"
#include "common_def.h"

// Implemetation of required functions
void agent_init() {
	// update global variable
	initGlobalVariables();

	// init qtable, agent action states
	int actionMemSize = NUM_AGENT * sizeof(int);
	int qtableMemSize = DIMENSION * DIMENSION * NUM_ACTIONS * sizeof(float);

	CHECK(hipMalloc((void **)&d_action, actionMemSize));
	CHECK(hipMalloc((void **)&d_qtable, qtableMemSize));

	initAgents();
	initQTable();
}

void agent_clearaction() {
	initAgents();
}

float agent_adjustepsilon() {
	return decEpsilon();
}

// the pointer is pointing to memory in GPU
// need to return pointer to memory in GPU
short* agent_action(int2* cstate) {
	updateActions(cstate);
	return d_action;
}

void agent_update(int2* cstate, int2* nstate, float *rewards) {

}